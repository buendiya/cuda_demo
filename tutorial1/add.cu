
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <math.h>


__global__
void add(int n, float *x, float *y)
{
  // printf("num of block %d\n", gridDim.x);
  // printf("num of thread in each block %d\n", blockDim.x);
  // printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1<<20;
  float *x, *y;


  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the GPU
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;
  
  auto t1 = std::chrono::high_resolution_clock::now();
  
  add<<<numBlocks, blockSize>>>(N, x, y);

  // add<<<1, 256>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  
    auto t2 = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>( t2 - t1 ).count();
    std::cout << "takes: " << duration  << " milliseconds" << std::endl;

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);
  
  return 0;
}