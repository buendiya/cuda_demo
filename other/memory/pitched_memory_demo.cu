/*
 * pitched_memory_demo.cu
 *
 *  Created on: Oct 21, 2020
 *      Author: jingsz
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>

int main(int argc, char **argv)
{
	// device pointers.
	float *d_pitch;
	float *d_normal;

	// matrix size.
	size_t cols = 256;
	size_t rows = 16;

	size_t pitch = 0;

	// alloc the data form gpu memory.
	hipMallocPitch((void**)&d_pitch, &pitch, cols*sizeof(float), rows);
	hipMalloc((void**)(&d_normal), rows*cols*sizeof(float));

	// test the data address.
	fprintf(stdout, "pitch = %lu \n", pitch);
	fprintf(stdout, "row size(in bytes) = %.2f*128.\n", pitch/128.0f);
	fprintf(stdout, "the head address of d_pitch  mod 128 = %x.\n", (unsigned int)((size_t)d_pitch)%128);
	fprintf(stdout, "the head address of d_normal mod 128 = %x.\n", (unsigned int)((size_t)d_normal)%128);

	hipFree(d_pitch);
	hipFree(d_normal);

	getchar();
	return 0;
}



