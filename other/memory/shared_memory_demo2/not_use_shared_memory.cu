/*
 * not_use_shared_memory.cpp
 *
 *  Created on: Dec 14, 2020
 *      Author: jingsz
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <chrono>
#include <iostream>
#include <random>     // mt19937 and uniform_int_distribution
#include <algorithm>  // generate
#include <vector>     // vector
#include <iterator>   // begin, end, and ostream_iterator
#include <functional> // bind
#include <hip/hip_runtime_api.h>

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct {
    int width;
    int height;
    float* elements;
} Matrix;

// Thread block size
#define BLOCK_SIZE 16

// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
    // Load A and B to device memory
    Matrix d_A;
    d_A.width = A.width; d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(float);
    hipMalloc((void**)&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size,
               hipMemcpyHostToDevice);
    Matrix d_B;
    d_B.width = B.width; d_B.height = B.height;
    size = B.width * B.height * sizeof(float);
    hipMalloc((void**)&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size,
               hipMemcpyHostToDevice);

    // Allocate C in device memory
    Matrix d_C;
    d_C.width = C.width; d_C.height = C.height;
    size = C.width * C.height * sizeof(float);
    hipMalloc((void**)&d_C.elements, size);

    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

    // Read C from device memory
    hipMemcpy((void*)C.elements, (void*)d_C.elements, size,
               hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}

// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
    // Each thread computes one element of C
    // by accumulating results into Cvalue
    float Cvalue = 0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    for (int e = 0; e < A.width; ++e)
        Cvalue += A.elements[row * A.width + e]
                * B.elements[e * B.width + col];
    C.elements[row * C.width + col] = Cvalue;
}


std::vector<float> create_random_data(int n) {
  std::random_device r;
  std::seed_seq      seed{r(), r(), r(), r(), r(), r(), r(), r()};
  std::mt19937       eng(seed); // a source of random data

  std::uniform_real_distribution<float> dist;
  std::vector<float> v(n);

  generate(begin(v), end(v), bind(dist, eng));
  return v;
}


int main(int argc, char **argv) {

	int dim = 1024*4;
	Matrix a = {dim, dim, nullptr};
	Matrix b = {dim, dim, nullptr};
	Matrix c = {dim, dim, nullptr};
	a.elements = (float*)malloc(dim * dim * sizeof(float));
	b.elements = (float*)malloc(dim * dim * sizeof(float));
	c.elements = (float*)malloc(dim * dim * sizeof(float));

	std::vector<float> input = create_random_data(dim*dim);
	std::cout << input[0] << std::endl;
    auto t1 = std::chrono::high_resolution_clock::now();

	memcpy(a.elements, input.data(), dim * dim * sizeof(float));
	memcpy(b.elements, input.data(), dim * dim * sizeof(float));
	MatMul(a, b, c);

    auto t2 = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>( t2 - t1 ).count();
    std::cout << "takes: " << duration/1000.0  << " milliseconds" << std::endl;

	free(a.elements);
	free(b.elements);
	free(c.elements);
	printf("finish\n");
}

